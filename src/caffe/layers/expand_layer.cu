#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"


namespace caffe {


template <typename Dtype>
void ExpandLayer<Dtype>::Forward_gpu (const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (this->layer_param_.use_cpu()) {
    Forward_cpu(bottom, top);
    return;
  }
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* bottom_data = bottom[0]->cpu_data();
  int spatial_dim = expand_w_ * expand_h_;
  for (int n = 0; n < bottom[0]->num(); n++) {
    for (int c = 0; c < channels_; c++) {
      caffe_gpu_set(spatial_dim, *bottom_data, 
            top_data);
      bottom_data += bottom[0]->offset(0,1);
      top_data += top[0]->offset(0,1);
    }
  }
}

template <typename Dtype>
__global__ void ExpandBackward(const int nthreads, const Dtype* top_diff,
    const int channels, const int height, const int width, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int c = index % channels;
    int n = index / channels;
    Dtype sumval = 0;
    top_diff += (n * channels + c) * height * width;
    for (int i = 0; i < height*width; ++i) {
      sumval += top_diff[i];
    }
    bottom_diff[index] = sumval;
  } 
}

template <typename Dtype>
void ExpandLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (this->layer_param_.use_cpu()) {
    Backward_cpu(top, propagate_down, bottom);
    return;
  }
  if (!propagate_down[0])
    return;
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int nthreads = top[0]->num()*channels_;
  ExpandBackward<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top_diff, channels_, expand_h_, expand_w_, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(ExpandLayer);

} // end of namespace caffe
